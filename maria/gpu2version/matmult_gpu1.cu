extern "C"{


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ 
void
matmultgpu1(int m, int n, int k, double *A, double *B, double *C) {
    	
    int tid=blockIdx.x*blockDim.x+threadIdx.x;

    int i1,i2,i3;

    //printf("to thread einai %d\n",tid);
    
    for(i1 = 0; i1< m; i1++){
    	for(i2 = 0; i2 < n; i2++){    
             C[i1*n+i2]=0;        
	    for(i3 = 0; i3 < k; i3++){ 
		C[i1*n+i2]+=A[i1*k+i3]*B[i3*n+i2];
	     }
        }
    }
	
}



void matmult_gpu1(int m, int n, int k, double *A, double *B, double *C){

  double   *d_A,*d_B,*d_C;

  int sizeA=m * k *sizeof(double);
  int sizeB=k * n *sizeof(double);
  int sizeC=m * n *sizeof(double);
 

  //Alloc memory on the device
  hipMalloc((void**)&d_A,sizeA);
  hipMalloc((void**)&d_B,sizeB);
  hipMalloc((void**)&d_C,sizeC);

  hipMemcpy(d_A,A,sizeA,hipMemcpyHostToDevice);
  hipMemcpy(d_B,B,sizeB,hipMemcpyHostToDevice);

/*
   dim3 dimBlock;
   dimBlock.x=1;
   dimBlock.y=1;



   dim3 dimgGrid;
   dimgGrid.x=1;
   dimgGrid.y=1;
*/
   

//  matmultgpu1<<<dimgGrid,dimBlock>>>(m,n,k,d_A,d_B,d_C);
  matmultgpu1<<<1,1>>>(m,n,k,d_A,d_B,d_C);
  hipDeviceSynchronize();

  hipMemcpy(A,d_A,sizeA,hipMemcpyDeviceToHost);
  hipMemcpy(B,d_B,sizeB,hipMemcpyDeviceToHost);
  hipMemcpy(C,d_C,sizeC,hipMemcpyDeviceToHost);
  
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);



}
}

