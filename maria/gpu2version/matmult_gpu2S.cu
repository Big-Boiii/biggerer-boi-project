extern "C"{


#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCK_SIZE 1

__global__ 
void
matmultgpu2(int m, int n, int k, double *A, double *B, double *C) {
    	
 float Cvalue=0;

  int col=blockIdx.x*blockDim.x+threadIdx.x;
  int row=blockIdx.y*blockDim.y+threadIdx.y;
  
  int e;
  for(e=0;e<k;++e)
      Cvalue+=A[row*k+e]*B[e*n+col];
  C[row*n+col]=Cvalue;
	
}



void matmult_gpu2(int m, int n, int k, double *A, double *B, double *C){

  double   *h_A,*h_B,*h_C,*d_A,*d_B,*d_C;
   
    
    int sizeA=m * k *sizeof(double);
    int sizeB=k * n *sizeof(double);
    int sizeC=m * n *sizeof(double);
 

    

    hipHostMalloc((void**)&h_A,sizeA, hipHostMallocDefault);
    hipHostMalloc((void**)&h_B,sizeB, hipHostMallocDefault);
    hipHostMalloc((void**)&h_C,sizeC, hipHostMallocDefault);

 

  //Alloc memory on the device
  hipMalloc((void**)&d_A,sizeA);
  hipMalloc((void**)&d_B,sizeB);
  hipMalloc((void**)&d_C,sizeC);

  hipMemcpy(d_A,A,sizeA,hipMemcpyHostToDevice);
  hipMemcpy(d_B,B,sizeB,hipMemcpyHostToDevice);

/*
 dim3 dimBlock;
 dimBlock.x=1;
 dimBlock.y=1;



  dim3 dimgGrid;
  

   // dimgGrid.x=(int)ceil((float)n/dimBlock.x);
   // dimgGrid.y=(int)ceil((float)m/dimBlock.y);
  dimgGrid.x=(int)ceil((float)(n+dimBlock.x-1)/dimBlock.x);
  dimgGrid.y=(int)ceil((float)(m+dimBlock.y-1)/dimBlock.y);
   	
 // dimgGrid.x=n;
//  dimgGrid.y=m;

  matmultgpu2<<<dimgGrid,dimBlock>>>(m,n,k,d_A,d_B,d_C);
*/

  dim3 numOfThreadsPerBlock;
  numOfThreadsPerBlock.x=BLOCK_SIZE;
  numOfThreadsPerBlock.y=BLOCK_SIZE;



  dim3 numOfBlocks;
  //numOfBlocks.x=(int)ceil((float)(n-numOfThreadsPerBlock.x-1)/numOfThreadsPerBlock.x);
  //numOfBlocks.y=(int)ceil((float)(m-numOfThreadsPerBlock.y-1)/numOfThreadsPerBlock.y);

/*
 numOfBlocks.x=(n+numOfThreadsPerBlock.x-1)/(numOfThreadsPerBlock.x);
  numOfBlocks.y=(m+numOfThreadsPerBlock.x-1)/(numOfThreadsPerBlock.y);

*/

 numOfBlocks.x=(n)/(numOfThreadsPerBlock.x);
  numOfBlocks.y=(m)/(numOfThreadsPerBlock.y);
  matmultgpu2<<<numOfBlocks,numOfThreadsPerBlock>>>(m,n,k,d_A,d_B,d_C);
  hipDeviceSynchronize();

  hipMemcpy(A,d_A,sizeA,hipMemcpyDeviceToHost);
  hipMemcpy(B,d_B,sizeB,hipMemcpyDeviceToHost);
  hipMemcpy(C,d_C,sizeC,hipMemcpyDeviceToHost);
  
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);



}
}

