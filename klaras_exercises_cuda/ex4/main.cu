#include <stdio.h>
#include <stdlib.h>
#include "mxv.h"

void print_matrix(int n, int k, double **C){
    for (int i=0; i<n; i++){
        for (int j=0; j<k; j++){
            printf("%lf \t", C[i][j]);
        }
        printf("\n");
    }
}

void print_vector(int m, double*B){
	for (int i=0; i<m; i++){
		printf("%lf \n", B[i]);
	}
}


double * allocate_vector(int m){
	double *A;

	A = (double*)malloc(m * sizeof(double *));	
	return A;
}


void assign_vector(double* vec, int m){
	for (int i=0; i<m; i++)
	{
		vec[i]=2.;
	}
}


int main(int argc, char* argv[]){
	int m =3;
	int n=2;

	double *A=allocate_vector(m*n); //A has n rows, m columns
	assign_vector(A, m*n);

	double *B=allocate_vector(n);
	assign_vector(B, n);

	double *C=allocate_vector(m);

	double *A_d;
	hipMalloc((void**)&A_d, m*n*sizeof(double));
	hipMemcpy(A_d,A,m*n*sizeof(double),hipMemcpyHostToDevice);

	double *B_d;
	hipMalloc((void**)&B_d, n*sizeof(double));
	hipMemcpy(B_d,B,n*sizeof(double),hipMemcpyHostToDevice);

	double *C_d;
	hipMalloc((void**)&C_d, m*sizeof(double));

	dim3 block_size(1);
	dim3 grid_size(m/block_size.x);
	mxv<<<block_size,grid_size>>>(m,n,A_d,B_d,C_d);
	hipDeviceSynchronize();

	hipMemcpy(C,C_d,m*sizeof(double),hipMemcpyDeviceToHost);
	print_vector(m,C);
	
}

