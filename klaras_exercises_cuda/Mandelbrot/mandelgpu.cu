
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void mandelgpu(int disp_width, int disp_height, int *array, int max_iter) {

    double 	scale_real, scale_imag;
    double 	x, y, u, v, u2, v2;
    int 	row,column, iter;
    column = threadIdx.y + blockIdx.y*blockDim.y;
    row= threadIdx.x + blockIdx.x*blockDim.x;	     

    scale_real = 3.5 / (double)disp_width;
    scale_imag = 3.5 / (double)disp_height;

    x = (column * scale_real) - 2.25; 
    y = (row * scale_imag) - 1.75; 

    u    = 0.0;
    v    = 0.0;
    u2   = 0.0;
    v2   = 0.0;
    iter = 0;

    while ( u2 + v2 < 4.0 &&  iter < max_iter ) {
	v = 2 * v * u + y;
	u = u2 - v2 + x;
	u2 = u*u;
	v2 = v*v;
	iter = iter + 1;
	}

    // if we exceed max_iter, reset to zero
    iter = iter == max_iter ? 0 : iter;
    //printf("This is Thread %d and Block %d. \n The value for this Cell is %d. \n", column, row, iter);
    array[row*disp_height + column] = iter;
}
