#include <stdio.h>
#include <stdlib.h>
#include "mandel.h"
#include "mandelgpu.h"
#include "writepng.h"
#include <omp.h>



int
main(int argc, char *argv[]) {

    int   width, height;
    int	  max_iter;
    int   *image_d;
    int   *image_h;

    width    = 5000;
    height   = 5000;
    max_iter = 400;

    // command line argument sets the dimensions of the image
    if ( argc == 2 ) width = height = atoi(argv[1]);

    
    image_h = (int *)malloc( width * height * sizeof(int));
    hipMalloc((void**)&image_d, width*height*sizeof(int));
    if ( image_h == NULL ) {
       fprintf(stderr, "memory allocation failed!\n");
       return(1);
    }
    //launch kernel
    dim3 block_size(10, 10);
    dim3 grid_size(width/block_size.x, height/ block_size.y);
    mandelgpu<<<grid_size,block_size>>>(width, height, image_d, max_iter);
    //mandel(width,height,image_h,max_iter);
    hipDeviceSynchronize();
    //copy results back
    hipMemcpy(image_h,image_d,width*height*sizeof(int), hipMemcpyDeviceToHost);
    writepng("mandelbrot.png", image_h, width, height);

    //cleanup results:
    hipFree(image_d);
    free(image_h);

    return(0);
}
