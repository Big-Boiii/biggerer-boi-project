
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void cuda_hello(){
//	printf("Hello\n");
	printf("Hello from Thread %d out of %d in block %d.\n  The ThreadID is %d of %d. \n", threadIdx.x +1, blockDim.x, blockIdx.x, threadIdx.x + (blockIdx.x*blockDim.x), 2*blockDim.x);



}

int main(){

	cuda_hello<<<8,2>>>();
	hipDeviceSynchronize();
	return 0;
}
