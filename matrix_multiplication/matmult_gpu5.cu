extern "C"
{


#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCK_SIZE 16

__global__ 
void
matmultgpu5(int m, int n, int k, double *A, double *B, double *C) {
    int blockRow = blockIdx.y,
        blockCol = blockIdx.x,
        row = threadIdx.y,
        col = threadIdx.x;
    int i, j, Asrow, Ascol, Bsrow, Bscol, Crow, Ccol;
    
    double Cvalue = 0.0;

    for (i = 0; i < (k / BLOCK_SIZE); i++) {
        
        __shared__ double As[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ double Bs[BLOCK_SIZE][BLOCK_SIZE];
        
        Asrow = blockRow * BLOCK_SIZE + row;
        Ascol = BLOCK_SIZE * i + col;
        
        As[row][col] = A[Asrow * k + Ascol];

        Bsrow = BLOCK_SIZE * i + row;
        Bscol = blockCol * BLOCK_SIZE + col;

        Bs[row][col] = B[Bsrow * n + Bscol];

        __syncthreads();

        for (j = 0; j < BLOCK_SIZE; j++) {
            Cvalue += As[row][j] * Bs[j][col];
        }
        __syncthreads();
    }

    Crow = blockRow * BLOCK_SIZE + row;
    Ccol = blockCol * BLOCK_SIZE + col;

    C[Crow *n + Ccol] = Cvalue;

}



void matmult_gpu5(int m, int n, int k, double *A, double *B, double *C){

    double *d_A, *d_B, *d_C;
  
  
    int sizeA = m * k *sizeof(double);
    int sizeB = k * n *sizeof(double);
    int sizeC = m * n *sizeof(double);
  
    // Allocate memory on the device
    hipMalloc((void**)&d_A, sizeA);
    hipMalloc((void**)&d_B, sizeB);
    hipMalloc((void**)&d_C, sizeC);
  
    // Copy the values over
    hipMemcpy(d_A, A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, sizeB, hipMemcpyHostToDevice);
  
    // Declare the number of threads
    dim3 numOfThreadsPerBlock;
    numOfThreadsPerBlock.x = BLOCK_SIZE;
    numOfThreadsPerBlock.y = BLOCK_SIZE;
  
    dim3 numOfBlocks;
    numOfBlocks.x = n/BLOCK_SIZE;
    numOfBlocks.y = m/BLOCK_SIZE;
  
  
    matmultgpu5<<<numOfBlocks, numOfThreadsPerBlock>>>(m, n, k, d_A, d_B, d_C);
    hipDeviceSynchronize();
  
    hipMemcpy(C, d_C, sizeC, hipMemcpyDeviceToHost);
  
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
}
  