extern "C"
{


#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCK_SIZE 16

__global__ 
void
matmultgpu2(int m, int n, int k, double *A, double *B, double *C) {
    	
  double Cvalue = 0.0;

  int col = blockIdx.x*blockDim.x+threadIdx.x;
  int row = blockIdx.y*blockDim.y+threadIdx.y;
  
  int e;

  if (row < m && col < n) {
    for(e=0;e<k;++e)
        Cvalue += A[row*k+e] * B[e*n+col];
        
    C[row*n+col] = Cvalue;
  }
	
}



void matmult_gpu2(int m, int n, int k, double *A, double *B, double *C){

  double *d_A, *d_B, *d_C;


  int sizeA = m * k *sizeof(double);
  int sizeB = k * n *sizeof(double);
  int sizeC = m * n *sizeof(double);

  // Allocate memory on the device
  hipMalloc((void**)&d_A, sizeA);
  hipMalloc((void**)&d_B, sizeB);
  hipMalloc((void**)&d_C, sizeC);

  // Copy the values over
  hipMemcpy(d_A, A, sizeA, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, sizeB, hipMemcpyHostToDevice);

  // Declare the number of threads
  dim3 numOfThreadsPerBlock;
  numOfThreadsPerBlock.x = BLOCK_SIZE;
  numOfThreadsPerBlock.y = BLOCK_SIZE;

  dim3 numOfBlocks;
  numOfBlocks.x = (n+numOfThreadsPerBlock.x-1)/(numOfThreadsPerBlock.x);
  numOfBlocks.y = (m+numOfThreadsPerBlock.x-1)/(numOfThreadsPerBlock.y);


  matmultgpu2<<<numOfBlocks, numOfThreadsPerBlock>>>(m, n, k, d_A, d_B, d_C);
  hipDeviceSynchronize();

  hipMemcpy(C, d_C, sizeC, hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}
}

