

#include <hip/hip_runtime.h>
#include <hipblas.h>

extern "C"
{

#include <stdio.h>


void matmult_gpulib(int m, int n, int k, double *A, double *B, double *C) {
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    const double alpha = 1.0, beta = 0.0;
    double *d_A, *d_B, *d_C;

    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
    }

    int sizeA = m * k *sizeof(double);
    int sizeB = k * n *sizeof(double);
    int sizeC = m * n *sizeof(double);
  
    // Allocate memory on the device
    hipMalloc((void**)&d_A, sizeA);
    hipMalloc((void**)&d_B, sizeB);
    hipMalloc((void**)&d_C, sizeC);
  
    // Copy the values over
    hipMemcpy(d_A, A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, sizeB, hipMemcpyHostToDevice);

    /*
    CUBLAS only reads matrices in column major form, and we hace A, B and C stored in row major.
    This will make CUBLAS understand our matrix as the transpose version if we input them normally.
    However, we want the output of C to be in rowmajor form too, so we need CUBLAS to calculate C^T.
    C^T = (AB)^T = B^T * A^T. Good news! We just need to swap the matrices out in the arguments to make
    CUBLAS output our C matrix in rowmajor form!
    */
    
    hipblasOperation_t transa = HIPBLAS_OP_N;
    hipblasOperation_t transb = HIPBLAS_OP_N;

    hipblasDgemm(handle, transa, transb, n, m, k, &alpha, d_B, n, d_A, k, &beta, d_C, n);
    hipDeviceSynchronize();

    hipMemcpy(C, d_C, sizeC, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
   
}
}